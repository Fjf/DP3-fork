#include "hip/hip_runtime.h"
// Copyright (C) 2023 ASTRON (Netherlands Institute for Radio Astronomy)
// SPDX-License-Identifier: GPL-3.0-or-later

#include "IterativeDiagonal.h"

#include <hip/hip_complex.h>
#include <hip/hip_math_constants.h>

#include "Common.h"
#include "Complex.h"
#include "MatrixComplex2x2.h"

#define BLOCK_SIZE 128

#define cudaCheckError() {                                      \
 hipError_t e=hipGetLastError();                                 \
 printf("testing cudaCheckError %s:%d\n",__FILE__,__LINE__); \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

template <bool Add>
__device__ void AddOrSubtract(size_t vis_index, size_t n_solutions,
                              const unsigned int* antenna_pairs,
                              const unsigned int* solution_map,
                              const hipDoubleComplex* solutions,
                              const cuM2x2FloatComplex* model,
                              const cuM2x2FloatComplex* residual_in,
                              cuM2x2FloatComplex* residual_out) {
  const uint32_t antenna_1 = antenna_pairs[vis_index * 2 + 0];
  const uint32_t antenna_2 = antenna_pairs[vis_index * 2 + 1];
  const size_t solution_index = solution_map[vis_index];
  const hipDoubleComplex* solution_1 =
      &solutions[(antenna_1 * n_solutions + solution_index) * 2];
  const hipDoubleComplex* solution_2 =
      &solutions[(antenna_2 * n_solutions + solution_index) * 2];

  const hipFloatComplex solution_1_0 = hipComplexDoubleToFloat(solution_1[0]);
  const hipFloatComplex solution_1_1 = hipComplexDoubleToFloat(solution_1[1]);
  const hipFloatComplex solution_2_0_conj =
      hipComplexDoubleToFloat(hipConj(solution_2[0]));
  const hipFloatComplex solution_2_1_conj =
      hipComplexDoubleToFloat(hipConj(solution_2[1]));

  const cuM2x2FloatComplex contribution(
      hipCmulf(hipCmulf(solution_1_0, model[vis_index][0]), solution_2_0_conj),
      hipCmulf(hipCmulf(solution_1_0, model[vis_index][1]), solution_2_1_conj),
      hipCmulf(hipCmulf(solution_1_1, model[vis_index][2]), solution_2_0_conj),
      hipCmulf(hipCmulf(solution_1_1, model[vis_index][3]), solution_2_1_conj));

  if (Add) {
    residual_out[vis_index] = residual_in[vis_index] + contribution;
  } else {
    residual_out[vis_index] = residual_in[vis_index] - contribution;
  }
}

__device__ void SolveDirection(size_t vis_index, size_t n_visibilities,
                               size_t n_direction_solutions, size_t n_solutions,
                               const unsigned int* antenna_pairs,
                               const unsigned int* solution_map,
                               const hipDoubleComplex* solutions,
                               const cuM2x2FloatComplex* model,
                               const cuM2x2FloatComplex* residual,
                               hipFloatComplex* numerator, float* denominator) {
  // Load correct variables to compute on.
  const size_t antenna_1 = antenna_pairs[vis_index * 2];
  const size_t antenna_2 = antenna_pairs[vis_index * 2 + 1];
  const size_t solution_index = solution_map[vis_index];

  const hipDoubleComplex* solution_antenna_1 =
      &solutions[(antenna_1 * n_solutions + solution_index) * 2];
  const hipDoubleComplex* solution_antenna_2 =
      &solutions[(antenna_2 * n_solutions + solution_index) * 2];

  const size_t rel_solution_index = solution_index - solution_map[0];

  // Calculate the contribution of this baseline for both antennas
  // For antenna2,
  // data_ba = data_ab^H, etc., therefore, numerator and denominator
  // become:
  // - num = data_ab^H * solutions_a * model_ab
  // - den = norm(model_ab^H * solutions_a)
  for (size_t i = 0; i < 2; i++) {
    const size_t antenna = antenna_pairs[vis_index * 2 + i];

    cuM2x2FloatComplex result;
    cuM2x2FloatComplex changed_model;

    if (i == 0) {
      const cuM2x2FloatComplexDiagonal solution(
          make_hipFloatComplex(solution_antenna_2[0].x, solution_antenna_2[0].y),
          make_hipFloatComplex(solution_antenna_2[1].x,
                              solution_antenna_2[1].y));
      changed_model = solution * hipConj(model[vis_index]);
      result = residual[vis_index] * changed_model;
    } else {
      const cuM2x2FloatComplexDiagonal solution(
          make_hipFloatComplex(solution_antenna_1[0].x, solution_antenna_1[0].y),
          make_hipFloatComplex(solution_antenna_1[1].x,
                              solution_antenna_1[1].y));
      changed_model = solution * model[vis_index];
      result = hipConj(residual[vis_index]) * changed_model;
    }

    const size_t full_solution_index =
        antenna * n_direction_solutions + rel_solution_index;

    // Atomic reduction into global memory
    atomicAdd(&numerator[full_solution_index * 2 + 0].x, result[0].x);
    atomicAdd(&numerator[full_solution_index * 2 + 0].y, result[0].y);
    atomicAdd(&numerator[full_solution_index * 2 + 1].x, result[3].x);
    atomicAdd(&numerator[full_solution_index * 2 + 1].y, result[3].y);

    atomicAdd(&denominator[full_solution_index * 2],
              cuNorm(changed_model[0]) + cuNorm(changed_model[2]));
    atomicAdd(&denominator[full_solution_index * 2 + 1],
              cuNorm(changed_model[1]) + cuNorm(changed_model[3]));
  }
}

__global__ void SolveDirectionKernel(
    size_t n_visibilities, size_t n_direction_solutions, size_t n_solutions,
    const unsigned int* antenna_pairs, const unsigned int* solution_map,
    const hipDoubleComplex* solutions, const cuM2x2FloatComplex* model,
    const cuM2x2FloatComplex* residual_in, cuM2x2FloatComplex* residual_temp,
    hipFloatComplex* numerator, float* denominator) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  AddOrSubtract<true>(vis_index, n_solutions, antenna_pairs, solution_map,
                      solutions, model, residual_in, residual_temp);

  SolveDirection(vis_index, n_visibilities, n_direction_solutions, n_solutions,
                 antenna_pairs, solution_map, solutions, model, residual_temp,
                 numerator, denominator);
}

void LaunchSolveDirectionKernel(
    hipStream_t stream, size_t n_visibilities, size_t n_direction_solutions,
    size_t n_solutions, size_t direction, cu::DeviceMemory& antenna_pairs,
    cu::DeviceMemory& solution_map, cu::DeviceMemory& solutions,
    cu::DeviceMemory& model, cu::DeviceMemory& residual_in,
    cu::DeviceMemory& residual_temp, cu::DeviceMemory& numerator,
    cu::DeviceMemory& denominator) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  const size_t direction_offset = direction * n_visibilities;
  const unsigned int* solution_map_direction =
      Cast<const unsigned int>(solution_map) + direction_offset;
  const cuM2x2FloatComplex* model_direction =
      Cast<const cuM2x2FloatComplex>(model) + direction_offset;
  SolveDirectionKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_visibilities, n_direction_solutions, n_solutions,
      Cast<const unsigned int>(antenna_pairs), solution_map_direction,
      Cast<const hipDoubleComplex>(solutions), model_direction,
      Cast<const cuM2x2FloatComplex>(residual_in),
      Cast<cuM2x2FloatComplex>(residual_temp), Cast<hipFloatComplex>(numerator),
      Cast<float>(denominator));
}

__global__ void SubtractKernel(size_t n_directions, size_t n_visibilities,
                               size_t n_solutions,
                               const unsigned int* antenna_pairs,
                               const unsigned int* solution_map,
                               const hipDoubleComplex* solutions,
                               const hipFloatComplex* model,
                               cuM2x2FloatComplex* residual) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  for (size_t direction = 0; direction < n_directions; direction++) {
    const size_t direction_offset = direction * n_visibilities;
    const unsigned int* solution_map_direction =
        solution_map + direction_offset;
    const hipFloatComplex* model_direction = model + (4 * direction_offset);
    AddOrSubtract<false>(
        vis_index, n_solutions, antenna_pairs, solution_map_direction,
        solutions, reinterpret_cast<const cuM2x2FloatComplex*>(model_direction),
        residual, residual);  // in-place
  }
}

void LaunchSubtractKernel(hipStream_t stream, size_t n_directions,
                          size_t n_visibilities, size_t n_solutions,
                          cu::DeviceMemory& antenna_pairs,
                          cu::DeviceMemory& solution_map,
                          cu::DeviceMemory& solutions, cu::DeviceMemory& model,
                          cu::DeviceMemory& residual) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  SubtractKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_directions, n_visibilities, n_solutions,
      Cast<const unsigned int>(antenna_pairs),
      Cast<const unsigned int>(solution_map),
      Cast<const hipDoubleComplex>(solutions), Cast<const hipFloatComplex>(model),
      Cast<cuM2x2FloatComplex>(residual));
}

__global__ void SolveNextSolutionKernel(unsigned int n_antennas,
                                        unsigned int n_direction_solutions,
                                        const unsigned int n_solutions,
                                        const unsigned int* solution_map,
                                        const hipFloatComplex* numerator,
                                        const float* denominator,
                                        hipDoubleComplex* next_solutions) {
  const size_t antenna = blockIdx.x * blockDim.x + threadIdx.x;

  if (antenna >= n_antennas) {
    return;
  }

  for (size_t relative_solution = 0; relative_solution < n_direction_solutions;
       relative_solution++) {
    const size_t solution_index = relative_solution + solution_map[0];
    hipDoubleComplex* destination =
        &next_solutions[(antenna * n_solutions + solution_index) * 2];
    const size_t index = antenna * n_direction_solutions + relative_solution;

    for (size_t pol = 0; pol < 2; pol++) {
      if (denominator[index * 2 + pol] == 0.0) {
        destination[pol] = {HIP_NAN, HIP_NAN};
      } else {
        // The CPU code performs this compuation in double-precision,
        // however single-precision also seems sufficiently accurate.
        destination[pol] = {
            numerator[index * 2 + pol].x / denominator[index * 2 + pol],
            numerator[index * 2 + pol].y / denominator[index * 2 + pol]};
      }
    }
  }
}

void LaunchSolveNextSolutionKernel(
    hipStream_t stream, size_t n_antennas, size_t n_visibilities,
    size_t n_direction_solutions, size_t n_solutions, size_t direction,
    cu::DeviceMemory& antenna_pairs, cu::DeviceMemory& solution_map,
    cu::DeviceMemory& next_solutions, cu::DeviceMemory& numerator,
    cu::DeviceMemory& denominator) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_antennas + block_dim) / block_dim;

  const size_t direction_offset = direction * n_visibilities;
  const unsigned int* solution_map_direction =
      Cast<const unsigned int>(solution_map) + direction_offset;
  SolveNextSolutionKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_antennas, n_direction_solutions, n_solutions, solution_map_direction,
      Cast<const hipFloatComplex>(numerator), Cast<const float>(denominator),
      Cast<hipDoubleComplex>(next_solutions));
}

__global__ void StepKernel(const size_t n_visibilities,
                           const hipDoubleComplex* solutions,
                           hipDoubleComplex* next_solutions, bool phase_only,
                           double step_size) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  if (phase_only) {
    // In phase only mode, a step is made along the complex circle,
    // towards the shortest direction.
    double phase_from = cuCarg(solutions[vis_index]);
    double distance = cuCarg(next_solutions[vis_index]) - phase_from;
    if (distance > HIP_PI)
      distance = distance - 2.0 * HIP_PI;
    else if (distance < -HIP_PI)
      distance = distance + 2.0 * HIP_PI;

    next_solutions[vis_index] =
        cuCpolar(1.0, phase_from + step_size * distance);
  } else {
    next_solutions[vis_index] =
        hipCadd(hipCmul(solutions[vis_index], (1.0 - step_size)),
               hipCmul(next_solutions[vis_index], step_size));
  }
}

void LaunchStepKernel(hipStream_t stream, size_t n_visibilities,
                      cu::DeviceMemory& solutions,
                      cu::DeviceMemory& next_solutions, bool phase_only,
                      double step_size) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  StepKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_visibilities, Cast<const hipDoubleComplex>(solutions),
      Cast<hipDoubleComplex>(next_solutions), phase_only, step_size);
  cudaCheckError();
}
