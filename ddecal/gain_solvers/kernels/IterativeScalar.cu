#include "hip/hip_runtime.h"
// Copyright (C) 2023 ASTRON (Netherlands Institute for Radio Astronomy)
// SPDX-License-Identifier: GPL-3.0-or-later

#include "IterativeScalar.h"

#include <hip/hip_complex.h>
#include <hip/hip_math_constants.h>

#include "Common.h"
#include "Complex.h"
#include "MatrixComplex2x2.h"

#define BLOCK_SIZE 128

template <bool Add>
__device__ void AddOrSubtractScalar(size_t vis_index, size_t n_solutions,
                              const unsigned int* antenna_pairs,
                              const unsigned int* solution_map,
                              const hipDoubleComplex* solutions,
                              const cuM2x2FloatComplex* model,
                              const cuM2x2FloatComplex* residual_in,
                              cuM2x2FloatComplex* residual_out) {
  const uint32_t antenna_1 = antenna_pairs[vis_index * 2 + 0];
  const uint32_t antenna_2 = antenna_pairs[vis_index * 2 + 1];
  const size_t solution_index = solution_map[vis_index];
  const hipDoubleComplex solution_1 =
      solutions[antenna_1 * n_solutions + solution_index];
  const hipDoubleComplex solution_2 =
      solutions[antenna_2 * n_solutions + solution_index];

  const hipFloatComplex solution_1_val = hipComplexDoubleToFloat(solution_1);
  const hipFloatComplex solution_2_conj =
      hipComplexDoubleToFloat(hipConj(solution_2));

  const cuM2x2FloatComplex contribution(
      hipCmulf(hipCmulf(solution_1_val, model[vis_index][0]), solution_2_conj),
      make_hipFloatComplex(0.0f, 0.0f),
      make_hipFloatComplex(0.0f, 0.0f),
      hipCmulf(hipCmulf(solution_1_val, model[vis_index][3]), solution_2_conj));

  if (Add) {
    residual_out[vis_index] = residual_in[vis_index] + contribution;
  } else {
    residual_out[vis_index] = residual_in[vis_index] - contribution;
  }
}

__device__ void SolveScalarDirection(size_t vis_index, size_t n_visibilities,
                               size_t n_direction_solutions, size_t n_solutions,
                               const unsigned int* antenna_pairs,
                               const unsigned int* solution_map,
                               const hipDoubleComplex* solutions,
                               const cuM2x2FloatComplex* model,
                               const cuM2x2FloatComplex* residual,
                               hipFloatComplex* numerator, float* denominator) {
  // Load correct variables to compute on.
  const size_t antenna_1 = antenna_pairs[vis_index * 2];
  const size_t antenna_2 = antenna_pairs[vis_index * 2 + 1];
  const size_t solution_index = solution_map[vis_index];

  const hipDoubleComplex solution_antenna_1 =
      solutions[antenna_1 * n_solutions + solution_index];
  const hipDoubleComplex solution_antenna_2 =
      solutions[antenna_2 * n_solutions + solution_index];

  const size_t rel_solution_index = solution_index - solution_map[0];

  // Calculate the contribution of this baseline for both antennas
  // For antenna2,
  // data_ba = data_ab^H, etc., therefore, numerator and denominator
  // become:
  // - num = data_ab^H * solutions_a * model_ab
  // - den = norm(model_ab^H * solutions_a)
  for (size_t i = 0; i < 2; i++) {
    const size_t antenna = antenna_pairs[vis_index * 2 + i];

    cuM2x2FloatComplex result;
    cuM2x2FloatComplex changed_model;

    if (i == 0) {
      const hipFloatComplex solution_val = 
          make_hipFloatComplex(solution_antenna_2.x, solution_antenna_2.y);
      changed_model = cuM2x2FloatComplex(
          hipCmulf(solution_val, hipConjf(model[vis_index][0])),
          make_hipFloatComplex(0.0f, 0.0f),
          make_hipFloatComplex(0.0f, 0.0f),
          hipCmulf(solution_val, hipConjf(model[vis_index][3])));
      result = residual[vis_index] * changed_model;
    } else {
      const hipFloatComplex solution_val = 
          make_hipFloatComplex(solution_antenna_1.x, solution_antenna_1.y);
      changed_model = cuM2x2FloatComplex(
          hipCmulf(solution_val, model[vis_index][0]),
          make_hipFloatComplex(0.0f, 0.0f),
          make_hipFloatComplex(0.0f, 0.0f),
          hipCmulf(solution_val, model[vis_index][3]));
      result = hipConj(residual[vis_index]) * changed_model;
    }

    const size_t full_solution_index =
        antenna * n_direction_solutions + rel_solution_index;

    // Atomic reduction into global memory
    atomicAdd(&numerator[full_solution_index * 2 + 0].x, result[0].x);
    atomicAdd(&numerator[full_solution_index * 2 + 0].y, result[0].y);
    atomicAdd(&numerator[full_solution_index * 2 + 1].x, result[3].x);
    atomicAdd(&numerator[full_solution_index * 2 + 1].y, result[3].y);

    atomicAdd(&denominator[full_solution_index * 2],
              cuNorm(changed_model[0]) + cuNorm(changed_model[2]));
    atomicAdd(&denominator[full_solution_index * 2 + 1],
              cuNorm(changed_model[1]) + cuNorm(changed_model[3]));
  }
}

__global__ void SolveScalarDirectionKernel(
    size_t n_visibilities, size_t n_direction_solutions, size_t n_solutions,
    const unsigned int* antenna_pairs, const unsigned int* solution_map,
    const hipDoubleComplex* solutions, const cuM2x2FloatComplex* model,
    const cuM2x2FloatComplex* residual_in, cuM2x2FloatComplex* residual_temp,
    hipFloatComplex* numerator, float* denominator) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  AddOrSubtractScalar<true>(vis_index, n_solutions, antenna_pairs, solution_map,
                      solutions, model, residual_in, residual_temp);

  SolveScalarDirection(vis_index, n_visibilities, n_direction_solutions, n_solutions,
                 antenna_pairs, solution_map, solutions, model, residual_temp,
                 numerator, denominator);
}

void LaunchScalarSolveDirectionKernel(
    hipStream_t stream, size_t n_visibilities, size_t n_direction_solutions,
    size_t n_solutions, size_t direction, cu::DeviceMemory& antenna_pairs,
    cu::DeviceMemory& solution_map, cu::DeviceMemory& solutions,
    cu::DeviceMemory& model, cu::DeviceMemory& residual_in,
    cu::DeviceMemory& residual_temp, cu::DeviceMemory& numerator,
    cu::DeviceMemory& denominator) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  const size_t direction_offset = direction * n_visibilities;
  const unsigned int* solution_map_direction =
      Cast<const unsigned int>(solution_map) + direction_offset;
  const cuM2x2FloatComplex* model_direction =
      Cast<const cuM2x2FloatComplex>(model) + direction_offset;
  SolveScalarDirectionKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_visibilities, n_direction_solutions, n_solutions,
      Cast<const unsigned int>(antenna_pairs), solution_map_direction,
      Cast<const hipDoubleComplex>(solutions), model_direction,
      Cast<const cuM2x2FloatComplex>(residual_in),
      Cast<cuM2x2FloatComplex>(residual_temp), Cast<hipFloatComplex>(numerator),
      Cast<float>(denominator));
}

__global__ void SubtractScalarKernel(size_t n_directions, size_t n_visibilities,
                               size_t n_solutions,
                               const unsigned int* antenna_pairs,
                               const unsigned int* solution_map,
                               const hipDoubleComplex* solutions,
                               const hipFloatComplex* model,
                               cuM2x2FloatComplex* residual) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  for (size_t direction = 0; direction < n_directions; direction++) {
    const size_t direction_offset = direction * n_visibilities;
    const unsigned int* solution_map_direction =
        solution_map + direction_offset;
    const hipFloatComplex* model_direction = model + (4 * direction_offset);
    AddOrSubtractScalar<false>(
        vis_index, n_solutions, antenna_pairs, solution_map_direction,
        solutions, reinterpret_cast<const cuM2x2FloatComplex*>(model_direction),
        residual, residual);  // in-place
  }
}

void LaunchScalarSubtractKernel(hipStream_t stream, size_t n_directions,
                          size_t n_visibilities, size_t n_solutions,
                          cu::DeviceMemory& antenna_pairs,
                          cu::DeviceMemory& solution_map,
                          cu::DeviceMemory& solutions, cu::DeviceMemory& model,
                          cu::DeviceMemory& residual) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  SubtractScalarKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_directions, n_visibilities, n_solutions,
      Cast<const unsigned int>(antenna_pairs),
      Cast<const unsigned int>(solution_map),
      Cast<const hipDoubleComplex>(solutions), Cast<const hipFloatComplex>(model),
      Cast<cuM2x2FloatComplex>(residual));
}

__global__ void SolveNextScalarSolutionKernel(unsigned int n_antennas,
                                        unsigned int n_direction_solutions,
                                        const unsigned int n_solutions,
                                        const unsigned int* solution_map,
                                        const hipFloatComplex* numerator,
                                        const float* denominator,
                                        hipDoubleComplex* next_solutions) {
  const size_t antenna = blockIdx.x * blockDim.x + threadIdx.x;

  if (antenna >= n_antennas) {
    return;
  }

  for (size_t relative_solution = 0; relative_solution < n_direction_solutions;
       relative_solution++) {
    const size_t solution_index = relative_solution + solution_map[0];
    hipDoubleComplex* destination =
        &next_solutions[(antenna * n_solutions + solution_index) * 2];
    const size_t index = antenna * n_direction_solutions + relative_solution;

    for (size_t pol = 0; pol < 2; pol++) {
      if (denominator[index * 2 + pol] == 0.0) {
        destination[pol] = {HIP_NAN, HIP_NAN};
      } else {
        // The CPU code performs this compuation in double-precision,
        // however single-precision also seems sufficiently accurate.
        destination[pol] = {
            numerator[index * 2 + pol].x / denominator[index * 2 + pol],
            numerator[index * 2 + pol].y / denominator[index * 2 + pol]};
      }
    }
  }
}

void LaunchScalarSolveNextSolutionKernel(
    hipStream_t stream, size_t n_antennas, size_t n_visibilities,
    size_t n_direction_solutions, size_t n_solutions, size_t direction,
    cu::DeviceMemory& antenna_pairs, cu::DeviceMemory& solution_map,
    cu::DeviceMemory& next_solutions, cu::DeviceMemory& numerator,
    cu::DeviceMemory& denominator) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_antennas + block_dim) / block_dim;

  const size_t direction_offset = direction * n_visibilities;
  const unsigned int* solution_map_direction =
      Cast<const unsigned int>(solution_map) + direction_offset;
  SolveNextScalarSolutionKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_antennas, n_direction_solutions, n_solutions, solution_map_direction,
      Cast<const hipFloatComplex>(numerator), Cast<const float>(denominator),
      Cast<hipDoubleComplex>(next_solutions));
}

__global__ void StepScalarKernel(const size_t n_visibilities,
                           const hipDoubleComplex* solutions,
                           hipDoubleComplex* next_solutions, bool phase_only,
                           double step_size) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  if (phase_only) {
    // In phase only mode, a step is made along the complex circle,
    // towards the shortest direction.
    double phase_from = cuCarg(solutions[vis_index]);
    double distance = cuCarg(next_solutions[vis_index]) - phase_from;
    if (distance > HIP_PI)
      distance = distance - 2.0 * HIP_PI;
    else if (distance < -HIP_PI)
      distance = distance + 2.0 * HIP_PI;

    next_solutions[vis_index] =
        cuCpolar(1.0, phase_from + step_size * distance);
  } else {
    next_solutions[vis_index] =
        hipCadd(hipCmul(solutions[vis_index], (1.0 - step_size)),
               hipCmul(next_solutions[vis_index], step_size));
  }
}

void LaunchScalarStepKernel(hipStream_t stream, size_t n_visibilities,
                      cu::DeviceMemory& solutions,
                      cu::DeviceMemory& next_solutions, bool phase_only,
                      double step_size) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  StepScalarKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_visibilities, Cast<const hipDoubleComplex>(solutions),
      Cast<hipDoubleComplex>(next_solutions), phase_only, step_size);
}
