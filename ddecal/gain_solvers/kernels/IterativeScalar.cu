#include "hip/hip_runtime.h"
// Copyright (C) 2023 ASTRON (Netherlands Institute for Radio Astronomy)
// SPDX-License-Identifier: GPL-3.0-or-later

#include "IterativeScalar.h"

#include <hip/hip_complex.h>
#include <hip/hip_math_constants.h>

#include "Common.h"
#include "Complex.h"
#include "MatrixComplex2x2.h"

#include <iostream>

#define BLOCK_SIZE 128

#define cudaCheckError() {                                      \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

template <bool Add>
__device__ void AddOrSubtractScalar(size_t vis_index, size_t n_solutions, size_t n_antenna,
                              const unsigned int* solution_map,
                              const hipDoubleComplex* solutions,
                              const hipFloatComplex* model,
                              const hipFloatComplex* residual_in,
                              hipFloatComplex* residual_out) {
  // Compute triangular index for the antenna pair
  // The pattern is: for each antenna_2, iterate through antenna_1 < antenna
  const size_t local_vis_index = vis_index % (n_antenna * (n_antenna - 1) / 2);
  const size_t antenna_2 = static_cast<uint32_t>((1.0 + std::sqrt(1.0 + 8.0 * local_vis_index)) / 2.0);
  const size_t antenna_1 = local_vis_index - (antenna_2 * (antenna_2 - 1)) / 2;  

  const size_t solution_index = solution_map[vis_index];
  const hipDoubleComplex solution_1 =
      solutions[antenna_1 * n_solutions + solution_index];
  const hipDoubleComplex solution_2 =
      solutions[antenna_2 * n_solutions + solution_index];

  const hipFloatComplex solution_1_val = hipComplexDoubleToFloat(solution_1);
  const hipFloatComplex solution_2_conj =
      hipComplexDoubleToFloat(hipConj(solution_2));

  const hipFloatComplex contribution = hipCmulf(hipCmulf(model[vis_index], solution_1_val), solution_2_conj);

  if (Add) {
    residual_out[vis_index] = hipCaddf(residual_in[vis_index], contribution);
  } else {
    residual_out[vis_index] = hipCsubf(residual_in[vis_index], contribution);
  }
}

__device__ void SolveScalarDirection(size_t vis_index, size_t n_visibilities,
                               size_t n_direction_solutions, size_t n_solutions, size_t n_antenna,
                               const unsigned int* solution_map,
                               const hipDoubleComplex* solutions,
                               const hipFloatComplex* model,
                               const hipFloatComplex* residual,
                               hipFloatComplex* numerator, float* denominator) {
  // Load correct variables to compute on.
  // Derive antenna indices from vis_index (for verification)
  // The pattern is: for each antenna_2, iterate through antenna_1 < antenna_2
  // vis_index 0: (0,1), vis_index 1: (0,2), vis_index 2: (1,2), vis_index 3: (0,3), etc.
  const size_t local_vis_index = vis_index % (n_antenna * (n_antenna - 1) / 2);
  const size_t antenna_2 = static_cast<uint32_t>((1.0 + std::sqrt(1.0 + 8.0 * local_vis_index)) / 2.0);
  const size_t antenna_1 = local_vis_index - (antenna_2 * (antenna_2 - 1)) / 2;

  const size_t solution_index = solution_map[vis_index];


  const hipDoubleComplex solution_antenna_1 =
      solutions[antenna_1 * n_solutions + solution_index];
  const hipDoubleComplex solution_antenna_2 =
      solutions[antenna_2 * n_solutions + solution_index];

  const size_t rel_solution_index = solution_index - solution_map[0];



  // Calculate the contribution of this baseline for both antennas
  // For antenna2,
  // data_ba = data_ab^H, etc., therefore, numerator and denominator
  // become:
  // - num = data_ab^H * solutions_a * model_ab
  // - den = norm(model_ab^H * solutions_a)
  for (size_t i = 0; i < 2; i++) {
    // const size_t antenna = antenna_pairs[vis_index * 2 + i];
    const size_t antenna = (i == 0) ? antenna_1 : antenna_2;

    hipFloatComplex result;
    hipFloatComplex changed_model;

    if (i == 0) {
      const hipFloatComplex solution_val =
          make_hipFloatComplex(solution_antenna_2.x, solution_antenna_2.y);

      // For scalar solving, we can directly access the model data
      const hipFloatComplex scalar_model = model[vis_index];
      const hipFloatComplex scalar_conj_model = hipConjf(scalar_model);
      const hipFloatComplex scalar_result = hipCmulf(solution_val, scalar_conj_model);

      changed_model = scalar_result;

      result = hipCmulf(residual[vis_index], scalar_result);

    } else {
      const hipFloatComplex solution_val =
          make_hipFloatComplex(solution_antenna_1.x, solution_antenna_1.y);

      // For scalar solving, we can directly access the model data
      const hipFloatComplex scalar_model = model[vis_index];
      const hipFloatComplex scalar_result = hipCmulf(solution_val, scalar_model);

      changed_model = scalar_result;

       result = hipCmulf(hipConjf(residual[vis_index]), scalar_result);
    }

    const size_t full_solution_index =
        antenna * n_direction_solutions + rel_solution_index;
    atomicAdd(&numerator[full_solution_index * 2].x, result.x);
    atomicAdd(&numerator[full_solution_index * 2].y, result.y);

    atomicAdd(&denominator[full_solution_index * 2],
              hipCabsf(changed_model) * hipCabsf(changed_model));
  }
}

__global__ void SolveScalarDirectionKernel(
    size_t n_visibilities, size_t n_direction_solutions, size_t n_solutions, size_t n_antenna,
    const unsigned int* solution_map, const hipDoubleComplex* solutions, const hipFloatComplex* model,
    const hipFloatComplex* residual_in, hipFloatComplex* residual_temp,
    hipFloatComplex* numerator, float* denominator) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  // Use the direction-specific pointers (solution_map and model are already offset for the current direction)
  AddOrSubtractScalar<true>(vis_index, n_solutions, n_antenna, solution_map,
                      solutions, model, residual_in, residual_temp);
  SolveScalarDirection(vis_index, n_visibilities, n_direction_solutions, n_solutions, n_antenna,
                solution_map, solutions, model, residual_temp,
                 numerator, denominator);
}

void LaunchScalarSolveDirectionKernel(
    hipStream_t stream, size_t n_visibilities, size_t n_direction_solutions,
    size_t n_solutions, size_t n_antenna, size_t direction,
    cu::DeviceMemory& solution_map, cu::DeviceMemory& solutions,
    cu::DeviceMemory& model, cu::DeviceMemory& residual_in,
    cu::DeviceMemory& residual_temp, cu::DeviceMemory& numerator,
    cu::DeviceMemory& denominator) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  const size_t direction_offset = direction * n_visibilities;
  const unsigned int* solution_map_direction =
      Cast<const unsigned int>(solution_map) + direction_offset;
  const hipFloatComplex* model_direction =
      Cast<const hipFloatComplex>(model) + direction_offset;
  SolveScalarDirectionKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_visibilities, n_direction_solutions, n_solutions, n_antenna, solution_map_direction,
      Cast<const hipDoubleComplex>(solutions), model_direction,
      Cast<const hipFloatComplex>(residual_in),
      Cast<hipFloatComplex>(residual_temp), Cast<hipFloatComplex>(numerator),
      Cast<float>(denominator));

  cudaCheckError();
}

__global__ void SubtractScalarKernel(size_t n_directions, size_t n_visibilities,
                               size_t n_solutions, size_t n_antenna,
                               const unsigned int* solution_map,
                               const hipDoubleComplex* solutions,
                               const hipFloatComplex* model,
                               hipFloatComplex* residual) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }
  for (size_t direction = 0; direction < n_directions; direction++) {
    const size_t direction_offset = direction * n_visibilities;
    const unsigned int* solution_map_direction =
        solution_map + direction_offset;
    const hipFloatComplex* model_direction = model + direction_offset;
    AddOrSubtractScalar<false>(
        vis_index, n_solutions, n_antenna, solution_map_direction,
        solutions, model_direction,
        residual, residual);  // in-place
  }
}
// TODO: Error is here, fix please
void LaunchScalarSubtractKernel(hipStream_t stream, size_t n_directions,
                          size_t n_visibilities, size_t n_solutions, size_t n_antenna,
                          cu::DeviceMemory& solution_map,
                          cu::DeviceMemory& solutions, cu::DeviceMemory& model,
                          cu::DeviceMemory& residual) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;



  SubtractScalarKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_directions, n_visibilities, n_solutions, n_antenna,
      Cast<const unsigned int>(solution_map),
      Cast<const hipDoubleComplex>(solutions), Cast<const hipFloatComplex>(model),
      Cast<hipFloatComplex>(residual));
}

__global__ void SolveNextScalarSolutionKernel(unsigned int n_antennas,
                                        unsigned int n_direction_solutions,
                                        const unsigned int n_solutions,
                                        const unsigned int* solution_map,
                                        const hipFloatComplex* numerator,
                                        const float* denominator,
                                        hipDoubleComplex* next_solutions) {
  const size_t antenna = blockIdx.x * blockDim.x + threadIdx.x;

  if (antenna >= n_antennas) {
    return;
  }

  for (size_t relative_solution = 0; relative_solution < n_direction_solutions;
       relative_solution++) {
    const size_t solution_index = relative_solution + solution_map[0];


    const size_t dest_idx = antenna * n_solutions + solution_index;


    const size_t index = antenna * n_direction_solutions + relative_solution;

    // Print values being used
    if (denominator[index * 2] == 0.0) {
      next_solutions[dest_idx] = {HIP_NAN, HIP_NAN};
    } else {
      next_solutions[dest_idx] = {
          numerator[index * 2].x / denominator[index * 2],
          numerator[index * 2].y / denominator[index * 2]};
    }
  }
}

void LaunchScalarSolveNextSolutionKernel(
    hipStream_t stream, size_t n_antennas, size_t n_visibilities,
    size_t n_direction_solutions, size_t n_solutions, size_t direction,
    cu::DeviceMemory& solution_map, cu::DeviceMemory& next_solutions,
    cu::DeviceMemory& numerator, cu::DeviceMemory& denominator) {

  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_antennas + block_dim - 1) / block_dim;

  const size_t direction_offset = direction * n_visibilities;

  const unsigned int* solution_map_direction =
      Cast<const unsigned int>(solution_map) + direction_offset;

  SolveNextScalarSolutionKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_antennas, n_direction_solutions, n_solutions, solution_map_direction,
      Cast<const hipFloatComplex>(numerator), Cast<const float>(denominator),
      Cast<hipDoubleComplex>(next_solutions));
  cudaCheckError();
}

__global__ void StepScalarKernel(const size_t n_visibilities,
                           const hipDoubleComplex* solutions,
                           hipDoubleComplex* next_solutions, bool phase_only,
                           double step_size) {
  const size_t vis_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (vis_index >= n_visibilities) {
    return;
  }

  if (phase_only) {
    // In phase only mode, a step is made along the complex circle,
    // towards the shortest direction.
    double phase_from = cuCarg(solutions[vis_index]);
    double distance = cuCarg(next_solutions[vis_index]) - phase_from;
    if (distance > HIP_PI)
      distance = distance - 2.0 * HIP_PI;
    else if (distance < -HIP_PI)
      distance = distance + 2.0 * HIP_PI;

    next_solutions[vis_index] =
        cuCpolar(1.0, phase_from + step_size * distance);
  } else {
    next_solutions[vis_index] =
        hipCadd(hipCmul(solutions[vis_index], (1.0 - step_size)),
               hipCmul(next_solutions[vis_index], step_size));
  }
}

void LaunchScalarStepKernel(hipStream_t stream, size_t n_visibilities,
                      cu::DeviceMemory& solutions,
                      cu::DeviceMemory& next_solutions, bool phase_only,
                      double step_size) {
  const size_t block_dim = BLOCK_SIZE;
  const size_t grid_dim = (n_visibilities + block_dim) / block_dim;

  StepScalarKernel<<<grid_dim, block_dim, 0, stream>>>(
      n_visibilities, Cast<const hipDoubleComplex>(solutions),
      Cast<hipDoubleComplex>(next_solutions), phase_only, step_size);
  cudaCheckError();
}
